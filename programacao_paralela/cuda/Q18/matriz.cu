#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

typedef struct {
    float *A, *B, *C;
    int device_id;
    int start_row;
    int rows_per_gpu;
    int N;
} ThreadData;

__global__ void add_matrices(const float *A, const float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        int idx = row * width + col;
        C[idx] = A[idx] + B[idx];
    }
}

void *gpu_worker(void *arg) {
    ThreadData *data = (ThreadData *)arg;
    int N = data->N;

    hipSetDevice(data->device_id);

    int chunk_size = data->rows_per_gpu * N;
    size_t bytes = chunk_size * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, bytes);
    hipMalloc((void **)&d_B, bytes);
    hipMalloc((void **)&d_C, bytes);

    hipMemcpy(d_A, data->A + data->start_row * N, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, data->B + data->start_row * N, bytes, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((N + 15) / 16, (data->rows_per_gpu + 15) / 16);

    add_matrices<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(data->C + data->start_row * N, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return NULL;
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Uso: %s <tamanho_N>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    if (N <= 0) {
        printf("Valor de N inválido.\n");
        return 1;
    }

    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    if (num_gpus == 0) {
        printf("Nenhuma GPU disponível.\n");
        return 1;
    }

    size_t size = N * N * sizeof(float);
    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    for (int i = 0; i < N * N; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    pthread_t threads[num_gpus];
    ThreadData thread_data[num_gpus];

    int rows_processed = 0;
    for (int i = 0; i < num_gpus; i++) {
        int start_row = rows_processed;
        int rows_for_this_gpu = N / num_gpus;

        // Último pega o resto se N não for múltiplo de num_gpus
        if (i == num_gpus - 1)
            rows_for_this_gpu = N - start_row;

        thread_data[i] = (ThreadData){
            .A = A,
            .B = B,
            .C = C,
            .device_id = i,
            .start_row = start_row,
            .rows_per_gpu = rows_for_this_gpu,
            .N = N
        };

        pthread_create(&threads[i], NULL, gpu_worker, &thread_data[i]);
        rows_processed += rows_for_this_gpu;
    }

    for (int i = 0; i < num_gpus; i++) {
        pthread_join(threads[i], NULL);
    }

    printf("C[0] = %f\n", C[0]);
    printf("C[N*N-1] = %f\n", C[N * N - 1]);

    free(A);
    free(B);
    free(C);

    return 0;
}
