#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define THREADS_PER_BLOCK 256

__device__ __host__ double f(double x)
{
    return x * x + 4; 
}

__global__ void trap(const double a, const double h, const long long n, const int t, double *val_trap)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= t) return;

    long long trap_per_thread = n / t;
    double local_sum = 0.0;
    double start_x = a + thread_id * trap_per_thread * h;

    int start_index = (thread_id == 0) ? 1 : 0;

    for (int i = start_index; i < trap_per_thread; i++)
    {
        double x = start_x + i * h;
        local_sum += f(x);
    }

    atomicAdd(val_trap, local_sum);
}
int main(int argc, char *argv[])
{
    if (argc != 5)
    {
        printf("Uso correto: %s <qnt_threads> <qnt_trap> <a> <b>\n", argv[0]);
        exit(1);
    }

    int qnt_threads = atoi(argv[1]);
    long long count_trap = atoll(argv[2]);

    if (count_trap % qnt_threads != 0)
    {
        printf("Erro: A quantidade de trapézios (%lld) deve ser divisível pelo número de threads (%d).\n", count_trap, qnt_threads);
        exit(1);
    }

    double a = atof(argv[3]);
    double b = atof(argv[4]);
    double h = (b - a) / count_trap;
    double val = 0.5 * (f(a) + f(b));
    int blocks = (qnt_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    double *d_val_trap;
    hipMalloc((void **)&d_val_trap, sizeof(double));
    hipMemcpy(d_val_trap, &val, sizeof(double), hipMemcpyHostToDevice);

    trap<<<blocks, THREADS_PER_BLOCK>>>(a, h, count_trap, qnt_threads, d_val_trap);
    hipDeviceSynchronize();

    hipMemcpy(&val, d_val_trap, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_val_trap);
    val = val * h;
    printf("Resultado da integral: %lf\n", val);

    return 0;
}